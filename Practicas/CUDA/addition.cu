
#include <hip/hip_runtime.h>
#include<stdio.h>
#define N 10

__global__ void addvec(int *a, int *b, int *c)
{
 int tid=blockIdx.x; //manejar los datos a este índice
 if(tid<N)
   c[tid]=a[tid]+b[tid];
}

//función principal
int main(void){
  int a[N], b[N], c[N];
  int *dev_a, *dev_b, *dev_c;

  //asignar memoria en la GPU
  hipMalloc((void**)&dev_a,N*sizeof(int));
  hipMalloc((void**)&dev_b,N*sizeof(int));
  hipMalloc((void**)&dev_c,N*sizeof(int));

  //Llenar datos a los arreglos 'a' y 'b' en la CPU
  for(int i=0; i<N;i++){
    a[i]=i;
    b[i]=i+1;
  }

  //copiar el arreglo 'a' y 'b' en la GPU
  hipMemcpy(dev_a,a,N*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(dev_b,b,N*sizeof(int),hipMemcpyHostToDevice);

  //lanzar el kernel
  addvec<<<N,1>>>(dev_a,dev_b,dev_c);

  //copiar el arreglo 'c' de la GPU a la CPU
  hipMemcpy(c,dev_c,N*sizeof(int),hipMemcpyDeviceToHost);

  //Desplegar el resultado
  for(int i=0; i<N; i++){
    printf("%d+%d=%d\n",a[i],b[i],c[i]);
  }
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);
  return 0;
  
} 