
#include <hip/hip_runtime.h>
#include<stdio.h>
#define N 1000

__global__ void addvec(int *a, int *b, int *c)
{
 int tid=blockIdx.x; //manejar los datos a este índice
 if(tid<N)
   c[tid]=a[tid]+b[tid];
}

//función principal
int main(void){
  int a[N], b[N], c[N];
  int *dev_a, *dev_b, *dev_c;

  //asignar memoria en la GPU
  hipMalloc((void**)&dev_a,N*sizeof(int));
  hipMalloc((void**)&dev_b,N*sizeof(int));
  hipMalloc((void**)&dev_c,N*sizeof(int));

  //Llenar datos a los arreglos 'a' y 'b' en la CPU
  for(int i=0; i<N;i++){
    a[i]=i;
    b[i]=i+1;
  }

  //GPU Time
  hipEvent_t start, stop;
  float time;

  //copiar el arreglo 'a' y 'b' en la GPU
  hipMemcpy(dev_a,a,N*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(dev_b,b,N*sizeof(int),hipMemcpyHostToDevice);

  hipEventCreate(&start); 
  hipEventCreate(&stop);
  hipEventRecord(start,0);

  //lanzar el kernel
  addvec<<<N,1>>>(dev_a,dev_b,dev_c);

  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);
	
  printf("Time : %f ms\n",time);

  //copiar el arreglo 'c' de la GPU a la CPU
  hipMemcpy(c,dev_c,N*sizeof(int),hipMemcpyDeviceToHost);

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);
  return 0;
  
} 
