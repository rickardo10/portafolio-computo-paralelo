#include <stdio.h> 
#include <hip/hip_runtime.h> 
//#include 
void DisplayProperties( hipDeviceProp_t* pDeviceProp )
{
	if( !pDeviceProp )
		return;	
	printf( "\n**************************************");
	printf( "\nDevice Name \t - %s ", pDeviceProp->name );
	printf( "\n**************************************");
	printf( "\nTotal Global Memory\t\t -> %ld KB", pDeviceProp->totalGlobalMem/1024 );
	printf( "\nShared memory available per block \t -> %ld KB", pDeviceProp->sharedMemPerBlock/1024 );
	printf( "\nNumber of registers per thread block \t -> %d", pDeviceProp->regsPerBlock );
	printf( "\nWarp size in threads \t -> %d", pDeviceProp->warpSize );
	printf( "\nMemory Pitch \t -> %lu bytes", pDeviceProp->memPitch );
	printf( "\nMaximum threads per block \t -> %d", pDeviceProp->maxThreadsPerBlock );
	printf( "\nMaximum Thread Dimension (block) \t -> %d %d %d", pDeviceProp->maxThreadsDim[0], pDeviceProp->maxThreadsDim[1], pDeviceProp->maxThreadsDim[2] );
	printf( "\nMaximum Thread Dimension (grid) \t -> %d %d %d", pDeviceProp->maxGridSize[0], pDeviceProp->maxGridSize[1], pDeviceProp->maxGridSize[2] );
	printf( "\nTotal constant memory \t -> %lu bytes", pDeviceProp->totalConstMem );
	printf( "\nCUDA ver \t -> %d.%d", pDeviceProp->major, pDeviceProp->minor );
	printf( "\nClock rate \t -> %d KHz", pDeviceProp->clockRate );
	printf( "\nTexture Alignment \t -> %lu bytes", pDeviceProp->textureAlignment );
	printf( "\nDevice Overlap \t -> %s", pDeviceProp-> deviceOverlap?"Allowed":"Not Allowed" );
	printf( "\nNumber of Multi processors \t -> %d\n", pDeviceProp->multiProcessorCount );
}int main(void)
{
	hipDeviceProp_t deviceProp;
	int nDevCount = 0;	hipGetDeviceCount( &nDevCount );
	printf( "Total Device found: %d", nDevCount );
	for (int nDeviceIdx = 0; nDeviceIdx < nDevCount; ++nDeviceIdx )
	{
		memset( &deviceProp, 0, sizeof(deviceProp));
		if( hipSuccess == hipGetDeviceProperties(&deviceProp, nDeviceIdx))
			DisplayProperties( &deviceProp );
		else
			printf( "\n%s", hipGetErrorString(hipGetLastError()));
	}
}

