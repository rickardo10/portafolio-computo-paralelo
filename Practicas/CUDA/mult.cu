#include <stdio.h>
#include <stdlib.h> 
#include <hip/hip_runtime.h>


#define BLOCK_SIZE 16


// Matrixes Multiplcation (Global Memory)
__global__ void multiply_gm(float *C,float *A,float *B, int nrow,int ncol)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int idy = blockIdx.y * blockDim.y + threadIdx.y;

  int index=idy*ncol+idx;

  if (idy<nrow && idx<ncol){
    float sum=0.0f;
    for(int k=0;k<ncol;k++){
      sum+=A[idy*ncol+k]*B[k*ncol+idx];
    }	
    C[index] = sum;
  }	
}

int div_up(int a,int b){
  return(a/b + (a%b == 0 ? 0:1));
}

int main(int argc, char* argv[]){
  float *A_h,*B_h,*C_h; // Host matrixes
  float *A_d,*B_d,*C_d;  //Device matrixes
  int nrow = atoi(argv[1]);  // rows
  int ncol = nrow;  // cols
  float N=nrow*ncol;  // number of elements

  //GPU Time
  hipEvent_t start, stop;
  float time;
	
  size_t size=N * sizeof(float);

  A_h = (float *)malloc(size); 
  B_h = (float *)malloc(size);
  C_h = (float *)malloc(size);

  //Initializing Host matrixes
  for (int i=0; i<nrow; i++){
    for(int j=0;j<ncol;j++){
      A_h[i*ncol+j] = 1.0f;
      B_h[i*ncol+j] = 2.0f;	
      //A_h[i*ncol+j] = rand()/100.0f;
      //B_h[i*ncol+j] = rand()/100.0f;	
    }
  }

  /*
  printf("\nMatrix A:\n");
  for (int i=0; i<nrow; i++){
    for(int j=0;j<ncol;j++){
      printf("%.2f ", A_h[i*ncol+j]);
    }
    printf("\n");
  }
  printf("\n\nMatrix B:\n");
  for (int i=0; i<nrow; i++){
    for(int j=0;j<ncol;j++){
      printf("%.2f ", B_h[i*ncol+j]);
    }
    printf("\n");
  }
  */

  hipMalloc((void **) &A_d,size);
  hipMalloc((void **) &B_d,size);
  hipMalloc((void **) &C_d,size);

  // Host to Device transfer
  hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
  hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

  //Realizamos el c�lculo en el Device
  dim3 block_size(BLOCK_SIZE,BLOCK_SIZE);
  dim3 n_blocks(div_up(ncol,block_size.x),div_up(nrow,block_size.y)) ;

  hipEventCreate(&start); 
  hipEventCreate(&stop);
  hipEventRecord(start,0);

  multiply_gm<<< n_blocks, block_size >>> (C_d,A_d,B_d,nrow,ncol);
	
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);
  printf("\t Time : %f ms\n",time);

  // Result from Device to Host
  hipMemcpy(C_h, C_d, size,hipMemcpyDeviceToHost);

  /*
  //Results
  printf("\n\nMatrix C:\n");
  for (int i=0; i<nrow; i++){
    for(int j=0;j<ncol;j++){
      printf("%.2f ", C_h[i*ncol+j]);
    }
    printf("\n");
  }
  */

  system("sleep 1");
	
  free(A_h); 
  free(B_h); 
  free(C_h); 

  hipFree(A_d); 
  hipFree(B_d); 
  hipFree(C_d); 
  return 0;
}