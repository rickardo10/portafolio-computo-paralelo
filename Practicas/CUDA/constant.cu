#include <stdio.h>
#include <stdlib.h> 
#include <hip/hip_runtime.h>


__constant__ int datos[1024];

__global__ void kernel(int *d_dst) {
  int tId = threadIdx.x + blockIdx.x * blockDim.x;
  d_dst[tId] = datos[tId];
}

int main(int argc, char **argv) {

  int *d_datos, *h_datos;
  hipMalloc((void**)&d_datos, sizeof(int) * 1024);
  h_datos = (int *)malloc(sizeof(int) * 1024);
  
  int *test = new int[1024];

  memset(test, 0, sizeof(int) * 1024);

  for (int i = 0; i < 1024; i++) {
    test[i] = i;
  }
 
  //GPU Time
  hipEvent_t start, stop;
  float time;

  hipMemcpyToSymbol(HIP_SYMBOL(datos), test, sizeof(int) * 1024);

  hipEventCreate(&start); 
  hipEventCreate(&stop);
  hipEventRecord(start,0);

  // Kernel call
  kernel<<< 1, 1024 >>>(d_datos);
 
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);

  // Copying From Device to Host
  hipMemcpy(h_datos, d_datos, sizeof(int)*1024, hipMemcpyDeviceToHost);
   for (int i = 0; i < 1024; i++)
     printf("\n %d", h_datos[i]); 

   printf("\t Time : %f ms\n",time);

  hipEventDestroy(start);
  hipEventDestroy(stop); 
  free(test);
  hipFree(d_datos); 

  return 0;
}