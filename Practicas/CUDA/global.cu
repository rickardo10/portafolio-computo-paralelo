#include <stdio.h>
#include <stdlib.h> 
#include <hip/hip_runtime.h>


//__constant__ int datos[1024];

__global__ void kernel(int *d_dst, int *d_src) {
  int tId = threadIdx.x + blockIdx.x * blockDim.x;
  d_dst[tId] = d_src[tId];
}

int main(int argc, char **argv) {

  int *d_datos, *h_datos, *d_src;
  hipMalloc((void**)&d_datos, sizeof(int) * 1024);
  hipMalloc((void**)&d_src, sizeof(int) * 1024);
  h_datos = (int *)malloc(sizeof(int) * 1024);
  
  int *test = new int[1024];

  memset(test, 0, sizeof(int) * 1024);

  for (int i = 0; i < 1024; i++) {
    test[i] = i;
  }
 
  //GPU Time
  hipEvent_t start, stop;
  float time;

  hipMemcpy(d_src, d_datos, sizeof(int)*1024, hipMemcpyHostToDevice);

  hipEventCreate(&start); 
  hipEventCreate(&stop);
  hipEventRecord(start,0);

  // Kernel call
  kernel<<< 1, 1024 >>>(d_datos, d_src);
 
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);

  // Copying From Device to Host
  hipMemcpy(h_datos, d_datos, sizeof(int)*1024, hipMemcpyDeviceToHost);
   for (int i = 0; i < 1024; i++)
     printf("\n %d", h_datos[i]); 

   printf("\t Time : %f ms\n",time);

  hipEventDestroy(start);
  hipEventDestroy(stop); 
  free(test);
  hipFree(d_datos); 

  return 0;
}