
#include <hip/hip_runtime.h>
#include<stdio.h>
#define N 10

__global__ void addvec(int *a, int *b, int *c) {
  __shared__ int s[N];
        
  if(threadIdx.x<N) {
    s[threadIdx.x]=a[threadIdx.x]+b[threadIdx.x]; 
  }
  __syncthreads(); 
  c[threadIdx.x]=s[threadIdx.x]; 
}

//función principal
int main(void){
  int a[N], b[N], c[N];
  int *dev_a, *dev_b, *dev_c;

  //asignar memoria en la GPU
  hipMalloc((void**)&dev_a,N*sizeof(int));
  hipMalloc((void**)&dev_b,N*sizeof(int));
  hipMalloc((void**)&dev_c,N*sizeof(int));

  //Llenar datos a los arreglos 'a' y 'b' en la CPU
  for(int i=0; i<N;i++){
    a[i]=i;
    b[i]=i+1;
  }

  //copiar el arreglo 'a' y 'b' en la GPU
  hipMemcpy(dev_a,a,N*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(dev_b,b,N*sizeof(int),hipMemcpyHostToDevice);
  
  //GPU Time
  hipEvent_t start, stop;
  float time;

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start,0);

  //lanzar el kernel
  addvec<<<N,1>>>(dev_a,dev_b,dev_c);

  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);

  //copiar el arreglo 'c' de la GPU a la CPU
  hipMemcpy(c,dev_c,N*sizeof(int),hipMemcpyDeviceToHost);

  //Desplegar el resultado
  for(int i=0; i<N; i++){
    printf("%d+%d=%d\n",a[i],b[i],c[i]);
  }
  
  printf("\t Time : %f ms\n",time);

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);
  return 0;
  
} 